#include "activation.cuh"

Activation::Activation(hipdnnActivationMode_t mode, double coef, hipdnnNanPropagation_t reluNanOpt)
{
    hipdnnCreateActivationDescriptor(&_desc);
    hipdnnSetActivationDescriptor(_desc, mode, reluNanOpt, coef);
}

Activation::~Activation()
{
    hipdnnDestroyActivationDescriptor(_desc);
}

Tensor& Activation::forward(const Tensor& x, Tensor& y)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    y.alloc(x.shape());

    hipdnnStatus_t status = hipdnnActivationForward(Global::getCudnnHandle(),
        _desc,
        &alpha,
        x.desc(),
        x.device(),
        &beta,
        y.desc(),
        y.device());
    return y;
}

