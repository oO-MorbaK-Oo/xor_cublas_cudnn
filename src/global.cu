#include "global.cuh"

namespace Global {

    hipblasHandle_t _h_cublas = nullptr;
    hipdnnHandle_t _h_cudnn = nullptr;

    void initialize()
    {
        auto cudaStatus = hipSetDevice(0);

        hipblasCreate(&_h_cublas);

        auto cudnnStatus = hipdnnCreate(&_h_cudnn);
    }

    void shutdown()
    {
        hipdnnDestroy(_h_cudnn);
        hipblasDestroy(_h_cublas);
        hipDeviceReset();
    }

    hipblasHandle_t getCublasHandle()
    {
        return _h_cublas;
    }

    hipdnnHandle_t getCudnnHandle()
    {
        return _h_cudnn;
    }
}